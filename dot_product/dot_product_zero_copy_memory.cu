#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <time.h>

#include <hip/hip_cooperative_groups.h>

#define BLOCK_SIZE 128
#define NUMBER_OF_BLOCKS 256

hipEvent_t start;
hipEvent_t stop;
hipError_t cudaStatus;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
		///assert(result == hipSuccess);
		exit(0);
	}
#endif

	return result;
}

template<typename T>
int CheckMemory(T* ptr) {

	if (ptr == NULL) {
		printf("Error. Allocation was unsuccessful. \n");
		return 0;
	}
	else
		return 1;
}

__device__ void reduceBlock( float* shared_data,float* dev_out, int index)
{
	int tid = threadIdx.x;

	__syncthreads();

	// do reduction in shared memory
	for (unsigned int s = 1; s < blockDim.x; s *= 2)
	{

		if (tid % (2 * s) == 0)
		{
			shared_data[tid] += shared_data[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0)
	{

		dev_out[index] = shared_data[tid];
	}

}



__global__ void DotProduct(float* a, float* b, float* dev_out, int size)
{
	__shared__ float shared_data[BLOCK_SIZE];


	int index = threadIdx.x + blockIdx.x * blockDim.x;

	int tid = threadIdx.x;

	float temp = 0.0f;

	//multiply together each dot product pair and move onto the next pair
	while (index < size)
	{
		temp += a[index] * b[index];
		index += blockDim.x * gridDim.x;

	}

	shared_data[tid] = temp;

	reduceBlock(shared_data, dev_out, blockIdx.x);
}



__global__ void Reduction(float* g_idata, int N)
{
	__shared__ float shared_data[NUMBER_OF_BLOCKS];
	unsigned int gridSize = blockDim.x * 2 * gridDim.x;



	int tid = threadIdx.x;

	if (tid < N)
	{

		shared_data[tid] = g_idata[tid];


		reduceBlock(shared_data, g_idata, blockIdx.x);
	}
}



// Helper function for using CUDA to add vectors in parallel.
void DotProductCuda(float *dev_a, float *dev_b,float* dev_out, const int arraySize, const int num_blocks, const int threadsPerBlock)
{


	// Variable to store output result
	float h_out;

	checkCuda(hipEventCreate(&start));
	checkCuda(hipEventCreate(&stop));


	hipError_t cudaStatus;

	float elapsedTime = 0.0f;


	dim3 Db(num_blocks);
	dim3 Dg(threadsPerBlock);

	hipEventRecord(start, 0);

	DotProduct << < Db, Dg >> > (dev_a, dev_b, dev_out, arraySize);

	checkCuda(hipEventRecord(stop, 0));
	checkCuda(hipEventSynchronize(stop)); //wait for the event to be executed!
	checkCuda(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time taken for DotProduct = %3.6fms\n", elapsedTime);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "DotProduct launch failed: %s\n", hipGetErrorString(cudaStatus));

	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	checkCuda(hipDeviceSynchronize());


	hipEventRecord(start, 0);

	Reduction << < 1, num_blocks >> > ( dev_out, arraySize);

	checkCuda(hipEventRecord(stop, 0));
	checkCuda(hipEventSynchronize(stop)); //wait for the event to be executed!
	checkCuda(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Time taken for Reduction = %3.6fms\n", elapsedTime);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Reduction launch failed: %s\n", hipGetErrorString(cudaStatus));

	}


	checkCuda(hipMemcpy(&h_out, &dev_out[0], sizeof(float), hipMemcpyDeviceToHost));

	//Print Result of Dot Product
	printf("\nResult = %3.6f\n", h_out);
}

float* InitData(float* a, float initVal, int size)
{

	for (int i = 0; i < size; i++)
	{

		a[i] = initVal;
	}
	return a;
}

int CalculateNumberOfBlocks(int arraySize)
{
	int num_blocks = NUMBER_OF_BLOCKS;
	if (arraySize < (NUMBER_OF_BLOCKS * BLOCK_SIZE))
	{
		num_blocks = (arraySize + (BLOCK_SIZE - 1)) / BLOCK_SIZE;
	}
	return num_blocks;
}


int main()
{


	hipDeviceProp_t prop;

	int device;

	checkCuda(hipGetDevice(&device));
	checkCuda(hipGetDeviceProperties(&prop, device));

	if (!prop.canMapHostMemory)
	{
		printf("Device cannot map host memory.\n");
		return 0;
	}

	checkCuda(hipSetDevice(0));
	checkCuda(hipSetDeviceFlags(hipDeviceMapHost));


	checkCuda(hipEventCreate(&start));
	checkCuda(hipEventCreate(&stop));


	const int arraySize = 65536;

	float elapsedTime = 0.0f;
	clock_t start_cpu, end_cpu;
	double cpu_time_used;


	// Declare pointers to host memory
	float* h_a;
	float* h_b;


	// Declare pointers to device memory
	float *dev_a = 0;
	float *dev_b = 0;
	float *dev_out = 0;


	checkCuda(hipHostAlloc((void**)&h_a, arraySize * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCuda(hipHostAlloc((void**)&h_b, arraySize * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));


	// Declare output array in device using hipMalloc because only read from device
	checkCuda(hipMalloc(&dev_out, NUMBER_OF_BLOCKS * sizeof(float)));

	// Retrieve device pointers in gpu memory space which map to host address space
	checkCuda(hipHostGetDevicePointer(&dev_a, h_a, 0));
	checkCuda(hipHostGetDevicePointer(&dev_b, h_b, 0));

	if (!CheckMemory<float>(h_a))
	{
		return 0;
	}
	if (!CheckMemory<float>(h_b))
	{
		return 0;
	}



	InitData(h_a, 1.0f, arraySize);
	InitData(h_b, 0.5f, arraySize);

	const int num_blocks = CalculateNumberOfBlocks(arraySize);

	const int threadsPerBlock = BLOCK_SIZE;


	DotProductCuda(dev_a, dev_b, dev_out, arraySize, num_blocks, threadsPerBlock);




	checkCuda(hipHostFree(h_a));
	checkCuda(hipHostFree(h_b));


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	checkCuda(hipDeviceReset());



	return 0;
}