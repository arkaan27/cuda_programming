#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void hello_cuda()
{

    printf("Hello Cuda World \n")
}

int main()
{
    hello_cuda << <1,1 >> > ();
    cudaDeviceSynhronize();

    hipDeviceReset();
    return 0;

}