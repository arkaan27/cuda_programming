#include "hip/hip_runtime.h"
#include <Windows.h>
#include <stdio.h>
#include <assert.h>


#define MBSIZE 1024 * 1024

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
        ///assert(result == hipSuccess);
    }
#endif
    return result;
}

void MemoryTest(int* host_array, int* dev_array, int n)
{


    checkCuda(hipMemcpy(dev_array, host_array, n * sizeof(int), hipMemcpyHostToDevice));


}



int main()
{
    int *host_arrayPaged, *host_arrayPinned;
    int *dev_array;

    float ms;

    int nMB = 256;

    int n = nMB * MBSIZE / sizeof(int);

    hipEvent_t startEvent, stopEvent;


    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));


    // Declare pinned memory
    checkCuda(hipHostAlloc((void**)&host_arrayPinned, n * sizeof(int), hipHostMallocDefault));

    host_arrayPaged = (int*)malloc(n * sizeof(int));

    checkCuda(hipMalloc((void**)&dev_array, n * sizeof(int)));


    checkCuda(hipEventRecord(startEvent, 0));

    MemoryTest(host_arrayPinned, dev_array, n);

    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));

    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
    //printf("%d, %f\n", i, 2 * nMB / ms);
    printf("Pinned memory (Gb/s): %f, total time = %3.6fms\n", 2 * nMB / ms, ms);

    checkCuda(hipEventRecord(startEvent, 0));

    MemoryTest(host_arrayPaged, dev_array, n);

    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));

    checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
    //printf("%d, %f\n", i, 2 * nMB / ms);
    printf("Paged memory (Gb/s): %f, total time = %3.6fms\n", 2 * nMB / ms, ms);


    return 0;
}