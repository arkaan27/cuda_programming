#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"


#include <Windows.h>
#include <stdio.h>

#define N 1024*1024
#define ARRAYSIZE N * 20


hipStream_t stream0;
hipStream_t stream1;

hipEvent_t kernel_start_event;
hipEvent_t kernel_stop_event;

hipEvent_t start, stop, sync_event0, sync_event1;

float elapsedTime;

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
		///assert(result == hipSuccess);
	}
#endif
	return result;
}

bool checkArray(int* a, int* b, int* c)
{

	for (int i = 0; i < ARRAYSIZE; i++)
	{

		float as = (a[i]);
		float bs = (b[i]);

		int tmp = c[i];

		if (tmp != (int)((as + bs) / 2))
		{
			return false;
		}

	}
	return true;
}

__global__ void kernel(int*a, int* b, int* c)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N)
	{
		//int idx1 = (idx + 1);// % 256;
		//int idx2 = (idx + 2);// % 256;


		float as = (a[idx]);
		float bs = (b[idx]);

		c[idx] = (as + bs) / 2;
	}
}

hipError_t  DefaultStreamExecution(int* dev_a0, int* dev_b0, int* dev_c0, int* host_a, int* host_b, int* host_c)
{
	hipError_t error;
	for (int i = 0; i < ARRAYSIZE; i += N)
	{

		checkCuda(hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice));
		checkCuda(hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice));

		kernel << <N / 256, 256 >> > (dev_a0, dev_b0, dev_c0);
		error = hipGetLastError();
		if (error != hipSuccess) {
			printf("kernel() failed to launch error = %d\n", error);
		}

		checkCuda(hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost));

	}



	return error;
}

hipError_t  ConcurrentStreamExecution(int* dev_a0, int* dev_b0, int* dev_c0, int* dev_a1, int* dev_b1, int* dev_c1, int* host_a, int* host_b, int* host_c)
{
	hipError_t error;
	for (int i = 0; i < ARRAYSIZE; i += N * 2)
	{

		checkCuda(hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));
		checkCuda(hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0));

		checkCuda(hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));
		checkCuda(hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1));
		kernel << <N / 256, 256, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);
		error = hipGetLastError();
		if (error != hipSuccess) {
			printf("kernel() failed to launch error = %d\n", error);
		}


		kernel << <N / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);
		error = hipGetLastError();
		if (error != hipSuccess) {
			printf("kernel() failed to launch error = %d\n", error);
		}
		checkCuda(hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0));
		checkCuda(hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1));

	}


	return error;
}


int main()
{
	hipError_t error;
	hipDeviceProp_t prop;

	int device;

	checkCuda(hipGetDevice(&device));
	checkCuda(hipGetDeviceProperties(&prop, device));

	if (!prop.deviceOverlap)
	{
		printf("Device does not handle overlaps\n");
		return 0;
	}



	checkCuda(hipEventCreate(&kernel_start_event));
	checkCuda(hipEventCreate(&kernel_stop_event));


	checkCuda(hipStreamCreate(&stream0));
	checkCuda(hipStreamCreate(&stream1));

	int *host_a, *host_b, *host_c;
	int *dev_a0, *dev_b0, *dev_c0;
	int *dev_a1, *dev_b1, *dev_c1;

	checkCuda(hipMalloc((void**)&dev_a0, N * sizeof(int)));
	checkCuda(hipMalloc((void**)&dev_b0, N * sizeof(int)));
	checkCuda(hipMalloc((void**)&dev_c0, N * sizeof(int)));

	checkCuda(hipMalloc((void**)&dev_a1, N * sizeof(int)));
	checkCuda(hipMalloc((void**)&dev_b1, N * sizeof(int)));
	checkCuda(hipMalloc((void**)&dev_c1, N * sizeof(int)));

	checkCuda(hipHostAlloc((void**)&host_a, ARRAYSIZE * sizeof(int), hipHostMallocDefault));
	checkCuda(hipHostAlloc((void**)&host_b, ARRAYSIZE * sizeof(int), hipHostMallocDefault));
	checkCuda(hipHostAlloc((void**)&host_c, ARRAYSIZE * sizeof(int), hipHostMallocDefault));


	for (int i = 0; i < ARRAYSIZE; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}


	checkCuda(hipEventRecord(kernel_start_event, 0));

	error = DefaultStreamExecution(dev_a0, dev_b0, dev_c0, host_a, host_b, host_c);

	if (error != hipSuccess) {
		printf("DefaultStreamExecution() failed to launch error = %d\n", error);
	}



	checkCuda(hipEventRecord(kernel_stop_event, 0));
	checkCuda(hipEventSynchronize(kernel_stop_event));

	checkCuda(hipEventElapsedTime(&elapsedTime, kernel_start_event, kernel_stop_event));

	printf("Serial Execution: Time taken: %3.1f ms\n", elapsedTime);

	if (!checkArray(host_a, host_b, host_c))
	{
		printf("Results don't match\n");
	}

	checkCuda(hipEventRecord(kernel_start_event, 0));



	error = ConcurrentStreamExecution(dev_a0, dev_b0, dev_c0, dev_a1, dev_b1, dev_c1, host_a, host_b, host_c);

	if (error != hipSuccess) {
		printf("ConcurrentStreamExecution() failed to launch error = %d\n", error);
	}

	checkCuda(hipStreamSynchronize(stream0));
	checkCuda(hipStreamSynchronize(stream1));

	checkCuda(hipEventRecord(kernel_stop_event, 0));
	checkCuda(hipEventSynchronize(kernel_stop_event));

	checkCuda(hipEventElapsedTime(&elapsedTime, kernel_start_event, kernel_stop_event));

	printf("Concurrent Execution: Time taken: %3.1f ms\n", elapsedTime);

	checkCuda(hipEventRecord(kernel_stop_event, 0));
	checkCuda(hipEventSynchronize(kernel_stop_event));

	checkCuda(hipEventElapsedTime(&elapsedTime, kernel_start_event, kernel_stop_event));

	if (!checkArray(host_a, host_b, host_c))
	{
		printf("Results don't match\n");
	}

	checkCuda(hipHostFree(host_a));
	checkCuda(hipHostFree(host_b));
	checkCuda(hipHostFree(host_c));
	checkCuda(hipFree(dev_a0));
	checkCuda(hipFree(dev_b0));
	checkCuda(hipFree(dev_c0));
	checkCuda(hipFree(dev_a1));
	checkCuda(hipFree(dev_b1));
	checkCuda(hipFree(dev_c1));

	checkCuda(hipStreamDestroy(stream0));
	checkCuda(hipStreamDestroy(stream1));

	return 0;
}