#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

#define RECURSION_DEPTH 10

__device__ int v = 0;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn", hipGetErrorString(result));
		///assert(result == hipSuccess);
	}
#endif
	return result;
}


__device__ void threadBlockDeviceSynchronize(void) {
	__syncthreads();
	if (threadIdx.x == 0)
		hipDeviceSynchronize();
	__syncthreads();
}


__global__ void recursiveKernel(int depth) {
	// up to depth 10



	if (depth == 10)
		return;


	// launch kernel on device
	if (threadIdx.x == 0) {
		printf("depth = %d\n", depth);
		// launch kernel on device
		hipStream_t s;
		hipStreamCreateWithFlags(&s, hipStreamNonBlocking);

		recursiveKernel << < 1, 8, 0, s >> > (depth + 1);

		//threadBlockDeviceSynchronize();

		if (depth == RECURSION_DEPTH-1) {

			printf("v = %d\n", v);
		}


		hipStreamDestroy(s);


	}

	atomicAdd(&v, 1);
}



int main()
{
	//parent_k << <1, 2 >> > ();
	int depth = 0;

	float elapsedTime = 0.0f;

	hipEvent_t startEvent, stopEvent;

	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));

	checkCuda(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 10));
	checkCuda(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 4096));

	checkCuda(hipEventRecord(startEvent, 0));

	recursiveKernel << <1, 8 >> > (depth);

	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	checkCuda(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));
	printf("total time = %3.6fms\n", elapsedTime);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("recursiveKernel() failed to launch error = %d\n", error);
	}


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	checkCuda(hipDeviceReset());


	return 0;
}